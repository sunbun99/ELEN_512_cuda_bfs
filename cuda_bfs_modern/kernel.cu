#include "hip/hip_runtime.h"
﻿
#ifndef CUDACC
#define CUDACC
#endif
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
//#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <conio.h>
#include <stdbool.h>
#include <ctype.h>
#include <list>

#include <windows.h>  


//#define FILE_NAME "4elt.graph.txt"
//#define NUM_NODES 15606
//#define NUM_EDGES 45878

//#define FILE_NAME "mdual.graph"
//#define NUM_NODES 258569
//#define NUM_EDGES 513132

#define FILE_NAME "citationCiteseer.graph"
#define NUM_NODES 268495
#define NUM_EDGES 1156647

#define PRINT_FRONT false
#define PRINT_COST false
#define PRINT_VISITED false

typedef struct
{
	int start;     // Index of first adjacent node in Ea	
	int length;    // Number of adjacent nodes 
} Node;

__global__ void CUDA_BFS_KERNEL(Node* Va, int* Ea, bool* Fa, bool* Xa, int* Ca, bool* done, int num_nodes)
{

	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id > num_nodes)
		//if (id > NUM_NODES)
		return;
	//*done = false;

	bool front = Fa[id];
	bool visted = Xa[id];
	int cost = Ca[id];

	if (Fa[id] == true && Xa[id] == false)
	{
		if(PRINT_FRONT)
			printf("%d ", id); //This printf gives the order of vertices in BFS	
		Fa[id] = false;
		Xa[id] = true;
		__syncthreads();
		// k = 0;
		//int i;
		int start = Va[id].start;
		int end = start + Va[id].length;
		for (int i = start; i < end; i++)
		{
			int nid = Ea[i];

			if (Xa[nid] == false)
			{
				Ca[nid] = Ca[id] + 1;
				Fa[nid] = true;
				*done = false;
			}

		}

	}

}

// The BFS frontier corresponds to all the nodes being processed at the current level.

//instead of creating a graph in the way that we do in the method seen below we should do CSR??
int main()
{
	FILE* myFile;
	myFile = fopen(FILE_NAME, "r");

	//myFile = fopen("test.graph", "r");
	
	if (myFile == NULL)
	{
		printf("Can't open the file\n");
		return 1;
	}

	//Node node[NUM_NODES];
	//int edges[NUM_EDGES];

	std::list<Node> nodes;

	Node n = { 0,0 };
	nodes.push_back(n); // add dummy element to help initialize list

	std::list<int> edges_;

	char buffer[10000];
	char* pbuff;
	char* tmpBuff;
	int value;
	int lineNum = 0;
	int edgeIndex = 0;
	int graph_stat = 0;
	int is_num = 0;
	int currLineNum = 0;
	int currLen = 0;
	int length = 0;

	while (1)
	{
		if (fgets(buffer, sizeof buffer, myFile) == 0)
			break;
		// printf("Line contains");
		pbuff = buffer;

		if (lineNum != 0) {
			//node[lineNum - 1].start = edgeIndex;
			//node[lineNum - 1].length = 0;
			currLineNum = lineNum - 1;
			currLen = 0;
		}

		while (1)
		{
			if (*pbuff == '\n')
				break;
			if (*pbuff == '\0')
				break;
			tmpBuff = pbuff;
			is_num = 0;
			while (*tmpBuff != '\0') {
				if (!isspace((unsigned char)*tmpBuff))
					is_num = 1;
				tmpBuff++;
			}
			if (is_num == 0)
				break;
			value = strtol(pbuff, &pbuff, 10);
			if (graph_stat == 0 && lineNum == 0)
			{
				printf("Creating graph of size %d \n", value);
				if (NUM_NODES != value)
				{
					printf("Node count not configured properly for loaded graph\n");
					return 1;
				}
				graph_stat = 1;
			}
			else if (lineNum == 0 && graph_stat == 1)
			{
				printf("%s", pbuff);
				printf("Expecting %d edges \n", value);
				if (NUM_EDGES != value)
				{
					printf("Edge count not configured properly for loaded graph\n");
					return 1;
				}
			}
			else
			{
				// printf("%d", value);
				/*
				node[lineNum - 1].length++;
				edges[edgeIndex] = value;*/
				edgeIndex++;
				currLen++;
				edges_.push_back(value);
				length++;
			}
			// if (value == 0)
			//     return 1;
		}
		if (lineNum != 0)
		{
			//Node n;
			n.length = length;
			n.start = nodes.back().start + nodes.back().length;
			nodes.push_back(n);
			length = 0;
		}
		lineNum++;
	}

	nodes.pop_front();  //remove dummy element from front

	Node* node;
	node = (Node*)malloc(nodes.size() * sizeof(Node));
	std::copy(nodes.begin(), nodes.end(), node);
	int* edges;
	edges = (int*)malloc(edges_.size() * sizeof(int));
	std::copy(edges_.begin(), edges_.end(), edges);



	//int edgesSize = 2 * NUM_NODES;
	/*
	int edges[NUM_NODES];

	node[0].start = 0;
	node[0].length = 2;

	node[1].start = 2;
	node[1].length = 1;

	node[2].start = 3;
	node[2].length = 1;

	node[3].start = 4;
	node[3].length = 1;

	node[4].start = 5;
	node[4].length = 0;

	edges[0] = 1;
	edges[1] = 2;
	edges[2] = 4;
	edges[3] = 3;
	edges[4] = 4; */

	//bool frontier[NUM_NODES] = { false };
	//bool visited[NUM_NODES] = { false };
	//int cost[NUM_NODES] = { 0 };

	bool* frontier;
	//frontier = (bool*)malloc(nodes.size() * sizeof(bool));
	frontier = (bool*)calloc(nodes.size(), sizeof(bool));

	bool* visited;
	//visited = (bool*)malloc(nodes.size() * sizeof(bool));
	visited = (bool*)calloc(nodes.size(), sizeof(bool));

	int* cost;
	//cost = (int*)malloc(nodes.size() * sizeof(int));
	cost = (int*)calloc(nodes.size(), sizeof(int));

	int const source = 0;
	frontier[source] = true;

	Node* Va;
	hipMalloc((void**)&Va, sizeof(Node) * nodes.size());
	hipMemcpy(Va, node, sizeof(Node) * nodes.size(), hipMemcpyHostToDevice);

	int* Ea;
	hipMalloc((void**)&Ea, sizeof(int) * edges_.size());
	hipMemcpy(Ea, edges, sizeof(int) * edges_.size(), hipMemcpyHostToDevice);

	bool* Fa;
	hipMalloc((void**)&Fa, sizeof(bool) * nodes.size());
	hipMemcpy(Fa, frontier, sizeof(bool) * nodes.size(), hipMemcpyHostToDevice);

	bool* Xa;
	hipMalloc((void**)&Xa, sizeof(bool) * nodes.size());
	hipMemcpy(Xa, visited, sizeof(bool) * nodes.size(), hipMemcpyHostToDevice);

	int* Ca;
	hipMalloc((void**)&Ca, sizeof(int) * nodes.size());
	hipMemcpy(Ca, cost, sizeof(int) * nodes.size(), hipMemcpyHostToDevice);



	//int num_blks = (512 / NUM_NODES)+1;
	int num_blks = (NUM_NODES / 512)+1;
	int threads = 512;



	bool done;
	bool* d_done;
	hipMalloc((void**)&d_done, sizeof(bool));
	printf("\n");
	int count = 0;

	LARGE_INTEGER frequency;        // ticks per second
	LARGE_INTEGER t1, t2;           // ticks
	double elapsedTime;

	// get ticks per second
	QueryPerformanceFrequency(&frequency);

	// start timer
	QueryPerformanceCounter(&t1);

	printf("Traversing graph \n");

	if(PRINT_FRONT)
		printf("Order: \n\n");
	do {
		count++;
		done = true;
		hipMemcpy(d_done, &done, sizeof(bool), hipMemcpyHostToDevice);
		CUDA_BFS_KERNEL <<< num_blks, threads >>> (Va, Ea, Fa, Xa, Ca, d_done, nodes.size());
		hipMemcpy(&done, d_done, sizeof(bool), hipMemcpyDeviceToHost);

	} while (!done);

	printf("Done \n\n");

	QueryPerformanceCounter(&t2);
	// compute and print the elapsed time in millisec
	elapsedTime = (t2.QuadPart - t1.QuadPart) * 1000.0 / frequency.QuadPart;
	printf("%f ms.\n", elapsedTime);




	hipMemcpy(cost, Ca, sizeof(int) * nodes.size(), hipMemcpyDeviceToHost);

	hipMemcpy(visited, Xa, sizeof(bool) * nodes.size(), hipMemcpyDeviceToHost);

	printf("Number of times the kernel is called : %d \n", count);



	//printf("\nCost: ");
	//for (int i = 0; i < nodes.size(); i++)
	//	printf("%d    ", cost[i]);
	//printf("\n");

	//printf("\nVisited: ");
	//for (int i = 0; i < nodes.size(); i++)
	//	printf("%d    ", visited[i]);
	//printf("\n");

	int missed = 0;

	for (int i = 0; i < nodes.size(); i++)
		if (!visited[i])
			missed++;

	printf("Missed nodes: %d\n", missed);

	hipFree(d_done);
	hipFree(Va);
	hipFree(Ea);
	hipFree(Fa);
	hipFree(Xa);
	hipFree(Ca);

	free(node);
	free(edges);
	_getch();

}
