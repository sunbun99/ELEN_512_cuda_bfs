#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <conio.h>
#define NUM_NODES 5

typedef struct
{
	int start;     // Index of first adjacent node in Ea	
	int length;    // Number of adjacent nodes 
} Node;

__global__ void CUDA_BFS_KERNEL(Node* Va, int* Ea, bool* Fa, bool* Xa, int* Ca, bool* done)
{

	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id > NUM_NODES)
		*done = false;


	if (Fa[id] == true && Xa[id] == false)
	{
		printf("%d ", id); //This printf gives the order of vertices in BFS	
		Fa[id] = false;
		Xa[id] = true;
		__syncthreads();
		int k = 0;
		int i;
		int start = Va[id].start;
		int end = start + Va[id].length;
		for (int i = start; i < end; i++)
		{
			int nid = Ea[i];

			if (Xa[nid] == false)
			{
				Ca[nid] = Ca[id] + 1;
				Fa[nid] = true;
				*done = false;
			}

		}

	}

}
__global__ void kernel_cuda_per_edge_basic(int *v_adj_from, int *v_adj_to, int num_edges, int *result, bool *still_running){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int num_threads = blockDim.x * gridDim.x;
	
	for (int e = 0; e < num_edges; e += num_threads){
		int edge = e + tid;
		if (edge < num_edges){
			int to_vertex = v_adj_to[edge];
			int new_len = result[v_adj_from[edge]] + 1;

			if (new_len < result[to_vertex]){
			result[to_vertex] = new_len;
			*still_running = true;
			}
		}
	}
}
// The BFS frontier corresponds to all the nodes being processed at the current level.


int main()
{




	Node node[NUM_NODES];


	//int edgesSize = 2 * NUM_NODES;
	int edges[NUM_NODES];

	node[0].start = 0;
	node[0].length = 2;

	node[1].start = 2;
	node[1].length = 1;

	node[2].start = 3;
	node[2].length = 1;

	node[3].start = 4;
	node[3].length = 1;

	node[4].start = 5;
	node[4].length = 0;

	edges[0] = 1;
	edges[1] = 2;
	edges[2] = 4;
	edges[3] = 3;
	edges[4] = 4;

	bool frontier[NUM_NODES] = { false };
	bool visited[NUM_NODES] = { false };
	int cost[NUM_NODES] = { 0 };

	int source = 0;
	frontier[source] = true;

	Node* Va;
	hipMalloc((void**)&Va, sizeof(Node) * NUM_NODES);
	hipMemcpy(Va, node, sizeof(Node) * NUM_NODES, hipMemcpyHostToDevice);

	int* Ea;
	hipMalloc((void**)&Ea, sizeof(Node) * NUM_NODES);
	hipMemcpy(Ea, edges, sizeof(Node) * NUM_NODES, hipMemcpyHostToDevice);

	bool* Fa;
	hipMalloc((void**)&Fa, sizeof(bool) * NUM_NODES);
	hipMemcpy(Fa, frontier, sizeof(bool) * NUM_NODES, hipMemcpyHostToDevice);

	bool* Xa;
	hipMalloc((void**)&Xa, sizeof(bool) * NUM_NODES);
	hipMemcpy(Xa, visited, sizeof(bool) * NUM_NODES, hipMemcpyHostToDevice);

	int* Ca;
	hipMalloc((void**)&Ca, sizeof(int) * NUM_NODES);
	hipMemcpy(Ca, cost, sizeof(int) * NUM_NODES, hipMemcpyHostToDevice);



	int num_blks = 1;
	int threads = 5;



	bool done;
	bool* d_done;
	hipMalloc((void**)&d_done, sizeof(bool));
	printf("\n\n");
	int count = 0;

	printf("Order: \n\n");
	do {
		count++;
		done = true;
		hipMemcpy(d_done, &done, sizeof(bool), hipMemcpyHostToDevice);
		CUDA_BFS_KERNEL << <num_blks, threads >> > (Va, Ea, Fa, Xa, Ca, d_done);
		hipMemcpy(&done, d_done, sizeof(bool), hipMemcpyDeviceToHost);

	} while (!done);




	hipMemcpy(cost, Ca, sizeof(int) * NUM_NODES, hipMemcpyDeviceToHost);

	printf("Number of times the kernel is called : %d \n", count);


	printf("\nCost: ");
	for (int i = 0; i < NUM_NODES; i++)
		printf("%d    ", cost[i]);
	printf("\n");
	_getch();

}
