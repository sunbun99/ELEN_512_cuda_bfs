#include "hip/hip_runtime.h"
﻿
#ifndef CUDACC
#define CUDACC
#endif
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
//#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <conio.h>
#include <stdbool.h>
#include <ctype.h>
#include <list>

#include <windows.h>  

#define NUM_NODES 15606
#define NUM_EDGES 45878

typedef struct
{
	int start;     // Index of first adjacent node in Ea	
	int length;    // Number of adjacent nodes 
} Node;

__global__ void CUDA_BFS_KERNEL(Node* Va, int* Ea, bool* Fa, bool* Xa, int* Ca, bool* done)
{

	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id > NUM_NODES)
		*done = false;


	if (Fa[id] == true && Xa[id] == false)
	{
		printf("%d ", id); //This printf gives the order of vertices in BFS	
		Fa[id] = false;
		Xa[id] = true;
		__syncthreads();
		// k = 0;
		//int i;
		int start = Va[id].start;
		int end = start + Va[id].length;
		for (int i = start; i < end; i++)
		{
			int nid = Ea[i];

			if (Xa[nid] == false)
			{
				Ca[nid] = Ca[id] + 1;
				Fa[nid] = true;
				*done = false;
			}

		}

	}

}
__global__ void kernel_cuda_simple(
    int *v_adj_list,
    int *v_adj_begin,
    int *v_adj_length,
    int num_vertices,
    int *result,
    bool *still_running)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;

    for (int v = 0; v < num_vertices; v += num_threads)
    {
        int vertex = v + tid;

        if (vertex < num_vertices)
        {
            for (int n = 0; n < v_adj_length[vertex]; n++)
            {
                int neighbor = v_adj_list[v_adj_begin[vertex] + n];

                if (result[neighbor] > result[vertex] + 1)
                {
                    result[neighbor] = result[vertex] + 1;
                    *still_running = true;
                }
            }
        }
    }
}

int bfs_cuda_simple(
    int *v_adj_list,
    int *v_adj_begin, 
    int *v_adj_length, 
    int num_vertices, 
    int num_edges,
    int start_vertex, 
    int *result)
{
    int *k_v_adj_list;
    int *k_v_adj_begin;
    int *k_v_adj_length;
    int *k_result;
    bool *k_still_running;

    int kernel_runs = 0;
    bool *still_running = new bool[1];
    fill_n(result, num_vertices, MAX_DIST);
    result[start_vertex] = 0;
    bool false_value = false;

    hipMalloc(&k_v_adj_list, sizeof(int) * num_edges);
    hipMalloc(&k_v_adj_begin, sizeof(int) * num_vertices);
    hipMalloc(&k_v_adj_length, sizeof(int) * num_vertices);
    hipMalloc(&k_result, sizeof(int) * num_vertices);
    hipMalloc(&k_still_running, sizeof(bool) * 1);

    hipMemcpy(k_v_adj_list, v_adj_list, sizeof(int) * num_edges, hipMemcpyHostToDevice);
    hipMemcpy(k_v_adj_begin, v_adj_begin, sizeof(int) * num_vertices, hipMemcpyHostToDevice);
    hipMemcpy(k_v_adj_length, v_adj_length, sizeof(int) * num_vertices, hipMemcpyHostToDevice);
    hipMemcpy(k_result, result, sizeof(int) * num_vertices, hipMemcpyHostToDevice);


    // --- START MEASURE TIME ---

    
    struct timeval t1, t2;
    gettimeofday(&t1, NULL);

    do
    {
        hipMemcpy(k_still_running, &false_value, sizeof(bool) * 1, hipMemcpyHostToDevice);

        kernel_cuda_simple<<<BLOCKS, THREADS>>>(
            k_v_adj_list, 
            k_v_adj_begin, 
            k_v_adj_length, 
            num_vertices, 
            k_result, 
            k_still_running);

        kernel_runs++;

        hipMemcpy(still_running, k_still_running, sizeof(bool) * 1, hipMemcpyDeviceToHost);
    } while (*still_running);

    hipDeviceSynchronize();

    gettimeofday(&t2, NULL);
    long long time = get_elapsed_time(&t1, &t2);

    if (report_time)
    {
        printf("%s,%i,%i,%i,%i,%lld\n", __FILE__, num_vertices, num_edges, BLOCKS, THREADS, time); 
    }


    // --- END MEASURE TIME ---


    hipMemcpy(result, k_result, sizeof(int) * num_vertices, hipMemcpyDeviceToHost);

    hipFree(k_v_adj_list);
    hipFree(k_v_adj_begin);
    hipFree(k_v_adj_length);
    hipFree(k_result);
    hipFree(k_still_running);

    // printf("%i kernel runs\n", kernel_runs);

    return time;
}
__global__ void kernel_cuda_per_edge_basic(int *v_adj_from, int *v_adj_to, int num_edges, int *result, bool *still_running){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int num_threads = blockDim.x * gridDim.x;
	
	for (int e = 0; e < num_edges; e += num_threads){
		int edge = e + tid;
		if (edge < num_edges){
			int to_vertex = v_adj_to[edge];
			int new_len = result[v_adj_from[edge]] + 1;

			if (new_len < result[to_vertex]){
			result[to_vertex] = new_len;
			*still_running = true;
			}
		}
	}
}

int bfs_cuda_per_edge_basic(
    int *v_adj_list,
    int *v_adj_begin, 
    int *v_adj_length, 
    int num_vertices, 
    int num_edges,
    int start_vertex, 
    int *result)
{
    // Convert data
    // TODO: Check if it is better to allocate only one array
    int *v_adj_from = new int[num_edges];
    int *v_adj_to = new int[num_edges];

    int next_index = 0;
    for (int i = 0; i < num_vertices; i++)
    {
        for (int j = v_adj_begin[i]; j < v_adj_length[i] + v_adj_begin[i]; j++)
        {
            v_adj_from[next_index] = i;
            v_adj_to[next_index++] = v_adj_list[j];
        }
    }

    int *k_v_adj_from;
    int *k_v_adj_to;
    int *k_result;
    bool *k_still_running;

    int kernel_runs = 0;

    fill_n(result, num_vertices, MAX_DIST);
    result[start_vertex] = 0;

    bool *still_running = new bool[1];

    hipMalloc(&k_v_adj_from, sizeof(int) * num_edges);
    hipMalloc(&k_v_adj_to, sizeof(int) * num_edges);
    hipMalloc(&k_result, sizeof(int) * num_vertices);
    hipMalloc(&k_still_running, sizeof(bool) * 1);

    hipMemcpy(k_v_adj_from, v_adj_from, sizeof(int) * num_edges, hipMemcpyHostToDevice);
    hipMemcpy(k_v_adj_to, v_adj_to, sizeof(int) * num_edges, hipMemcpyHostToDevice);
    hipMemcpy(k_result, result, sizeof(int) * num_vertices, hipMemcpyHostToDevice);


    // --- START MEASURE TIME ---


    struct timeval t1, t2;
    gettimeofday(&t1, NULL);

    do
    {
        *still_running = false;
        hipMemcpy(k_still_running, still_running, sizeof(bool) * 1, hipMemcpyHostToDevice);

        kernel_cuda_per_edge_basic<<<BLOCKS, THREADS>>>(
            k_v_adj_from, 
            k_v_adj_to, 
            num_edges, 
            k_result, 
            k_still_running);

        kernel_runs++;

        hipMemcpy(still_running, k_still_running, sizeof(bool) * 1, hipMemcpyDeviceToHost);
    } while (*still_running);

    hipDeviceSynchronize();

    gettimeofday(&t2, NULL);
    long long time = get_elapsed_time(&t1, &t2);

    if (report_time)
    {
        printf("%s,%i,%i,%i,%i,%lld\n", __FILE__, num_vertices, num_edges, BLOCKS, THREADS, time); 
    }


    // --- END MEASURE TIME ---


    hipMemcpy(result, k_result, sizeof(int) * num_vertices, hipMemcpyDeviceToHost);

    hipFree(k_v_adj_from);
    hipFree(k_v_adj_to);
    hipFree(k_result);
    hipFree(k_still_running);

    free(v_adj_from);
    free(v_adj_to);

    // printf("%i kernel runs\n", kernel_runs);

    return time;
}

// The BFS frontier corresponds to all the nodes being processed at the current level.

//instead of creating a graph in the way that we do in the method seen below we should do CSR??
int main()
{
	FILE* myFile;
	myFile = fopen("4elt.graph.txt", "r");
	// myFile = fopen("mdual.graph", "r");
	// myFile = fopen("citationCiteseer.graph", "r");

	//myFile = fopen("test.graph", "r");
	
	if (myFile == NULL)
	{
		printf("Can't open the file\n");
		return 1;
	}

	//Node node[NUM_NODES];
	//int edges[NUM_EDGES];

	std::list<Node> nodes;

	Node n = { 0,0 };
	nodes.push_back(n); // add dummy element to help initialize list

	std::list<int> edges_;

	char buffer[10000];
	char* pbuff;
	char* tmpBuff;
	int value;
	int lineNum = 0;
	int edgeIndex = 0;
	int graph_stat = 0;
	int is_num = 0;
	int currLineNum = 0;
	int currLen = 0;
	int length = 0;

	while (1)
	{
		if (fgets(buffer, sizeof buffer, myFile) == 0)
			break;
		// printf("Line contains");
		pbuff = buffer;

		if (lineNum != 0) {
			//node[lineNum - 1].start = edgeIndex;
			//node[lineNum - 1].length = 0;
			currLineNum = lineNum - 1;
			currLen = 0;
		}

		while (1)
		{
			if (*pbuff == '\n')
				break;
			if (*pbuff == '\0')
				break;
			tmpBuff = pbuff;
			is_num = 0;
			while (*tmpBuff != '\0') {
				if (!isspace((unsigned char)*tmpBuff))
					is_num = 1;
				tmpBuff++;
			}
			if (is_num == 0)
				break;
			value = strtol(pbuff, &pbuff, 10);
			if (graph_stat == 0 && lineNum == 0)
			{
				printf("Creating graph of size %d \n", value);
				if (NUM_NODES != value)
				{
					printf("Node count not configured properly for loaded graph\n");
					return 1;
				}
				graph_stat = 1;
			}
			else if (lineNum == 0 && graph_stat == 1)
			{
				printf("%s", pbuff);
				printf("Expecting %d edges \n", value);
				if (NUM_EDGES != value)
				{
					printf("Edge count not configured properly for loaded graph\n");
					return 1;
				}
			}
			else
			{
				// printf("%d", value);
				/*
				node[lineNum - 1].length++;
				edges[edgeIndex] = value;*/
				edgeIndex++;
				currLen++;
				edges_.push_back(value);
				length++;
			}
			// if (value == 0)
			//     return 1;
		}
		if (lineNum != 0)
		{
			//Node n;
			n.length = length;
			n.start = nodes.back().start + nodes.back().length;
			nodes.push_back(n);
			length = 0;
		}
		lineNum++;
	}

	nodes.pop_front();  //remove dummy element from front

	Node* node;
	node = (Node*)malloc(nodes.size() * sizeof(Node));
	std::copy(nodes.begin(), nodes.end(), node);
	int* edges;
	edges = (int*)malloc(edges_.size() * sizeof(int));
	std::copy(edges_.begin(), edges_.end(), edges);



	//int edgesSize = 2 * NUM_NODES;
	/*
	int edges[NUM_NODES];

	node[0].start = 0;
	node[0].length = 2;

	node[1].start = 2;
	node[1].length = 1;

	node[2].start = 3;
	node[2].length = 1;

	node[3].start = 4;
	node[3].length = 1;

	node[4].start = 5;
	node[4].length = 0;

	edges[0] = 1;
	edges[1] = 2;
	edges[2] = 4;
	edges[3] = 3;
	edges[4] = 4; */

	//bool frontier[NUM_NODES] = { false };
	//bool visited[NUM_NODES] = { false };
	//int cost[NUM_NODES] = { 0 };

	bool* frontier;
	//frontier = (bool*)malloc(nodes.size() * sizeof(bool));
	frontier = (bool*)calloc(nodes.size(), sizeof(bool));

	bool* visited;
	//visited = (bool*)malloc(nodes.size() * sizeof(bool));
	visited = (bool*)calloc(nodes.size(), sizeof(bool));

	int* cost;
	//cost = (int*)malloc(nodes.size() * sizeof(int));
	cost = (int*)calloc(nodes.size(), sizeof(int));

	int const source = 0;
	frontier[source] = true;

	Node* Va;
	hipMalloc((void**)&Va, sizeof(Node) * nodes.size());
	hipMemcpy(Va, node, sizeof(Node) * nodes.size(), hipMemcpyHostToDevice);

	int* Ea;
	hipMalloc((void**)&Ea, sizeof(Node) * nodes.size());
	hipMemcpy(Ea, edges, sizeof(Node) * nodes.size(), hipMemcpyHostToDevice);

	bool* Fa;
	hipMalloc((void**)&Fa, sizeof(bool) * nodes.size());
	hipMemcpy(Fa, frontier, sizeof(bool) * nodes.size(), hipMemcpyHostToDevice);

	bool* Xa;
	hipMalloc((void**)&Xa, sizeof(bool) * nodes.size());
	hipMemcpy(Xa, visited, sizeof(bool) * nodes.size(), hipMemcpyHostToDevice);

	int* Ca;
	hipMalloc((void**)&Ca, sizeof(int) * nodes.size());
	hipMemcpy(Ca, cost, sizeof(int) * nodes.size(), hipMemcpyHostToDevice);



	//int num_blks = (512 / NUM_NODES)+1;
	int num_blks = NUM_NODES / 512;
	int threads = 512;



	bool done;
	bool* d_done;
	hipMalloc((void**)&d_done, sizeof(bool));
	printf("\n\n");
	int count = 0;

	printf("Order: \n\n");
	do {
		count++;
		done = true;
		hipMemcpy(d_done, &done, sizeof(bool), hipMemcpyHostToDevice);
		CUDA_BFS_KERNEL <<< num_blks, threads >>> (Va, Ea, Fa, Xa, Ca, d_done);
		hipMemcpy(&done, d_done, sizeof(bool), hipMemcpyDeviceToHost);

	} while (!done);




	hipMemcpy(cost, Ca, sizeof(int) * nodes.size(), hipMemcpyDeviceToHost);

	printf("Number of times the kernel is called : %d \n", count);


	printf("\nCost: ");
	for (int i = 0; i < NUM_NODES; i++)
		printf("%d    ", cost[i]);
	printf("\n");
	_getch();

}
